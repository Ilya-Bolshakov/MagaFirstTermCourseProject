#include "hip/hip_runtime.h"
#include "Kernel.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
//#include "hip/device_functions.h"
__global__ void kernel(unsigned* src, int width,
	int height) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if ((x < width) && (y < height))
	{
		unsigned pixel = src[y * width + x];
		unsigned gray = ((pixel & 0x00FF0000 >> 0x10) +
			(pixel & 0x0000FF00 >> 0x8) + (pixel & 0x000000FF))
			/ 3;
		src[y * width + x] = 0xFF000000 | (gray << 0x10) |
			(gray << 0x8) | gray;
	}
}
int calc(unsigned* srcImage, int width, int height)
{
	unsigned* dev_srcImage;
	size_t size = sizeof(unsigned) * width * height;
	if (hipMalloc((void**)&dev_srcImage, size) !=
		hipError_t::hipSuccess) return -1;
	if (hipMemcpy(dev_srcImage, srcImage, size,
		hipMemcpyHostToDevice) != hipError_t::hipSuccess)
		return -1;
	// ������������ ���������� ����� �� ���� �����
	dim3 threads(128, 128);
	dim3 blocks((width + threads.x - 1) / threads.x,
		(height + threads.y - 1) / threads.y);
	kernel <<<threads, blocks>>> (dev_srcImage, width,
		height);
	hipError_t error = hipMemcpy(srcImage, dev_srcImage,
		size,
		hipMemcpyDeviceToHost);
	if (error != hipError_t::hipSuccess) return -1;
	hipFree(dev_srcImage);
	return 0;
}
