#include "hip/hip_runtime.h"
#include "CudaImage.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../MagaFirstTermCourseProject.FormsApp/ClusteredImage.h"
#include <iostream>

using namespace CommonTypes;


__global__ void kmeans_kernel(float* d_data, float* d_centroids, int* d_labels, int* k, int* countPixels) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < *countPixels) {
		double min_dist = sqrt((d_data[idx * 3] - d_centroids[0]) * (d_data[idx * 3] - d_centroids[0])
							   + (d_data[idx * 3 + 1] - d_centroids[1]) * (d_data[idx * 3 + 1] - d_centroids[1])
								   + (d_data[idx * 3 + 2] - d_centroids[2]) * (d_data[idx * 3 + 2] - d_centroids[2]));
		int minIndex = 0;

		for (int j = 1; j < *k; j++) {
			//float dist = 0.0f;

			double dist = sqrt((d_data[idx * 3] - d_centroids[j * 3]) * (d_data[idx * 3] - d_centroids[j * 3])
				+ (d_data[idx * 3 + 1] - d_centroids[j * 3 + 1]) * (d_data[idx * 3 + 1] - d_centroids[j * 3 + 1])
				+ (d_data[idx * 3 + 2] - d_centroids[j * 3 + 2]) * (d_data[idx * 3 + 2] - d_centroids[j * 3 + 2]));

			if (dist < min_dist) {
				min_dist = dist;
				minIndex = j;
			}
		}

		if (d_labels[idx] != minIndex) {
			d_labels[idx] = minIndex;
		}
	}
}



ClusteredImage calc(std::vector<Pixel>& pixels, int k)
{
	float* device_data;
	float* device_centroids;
	int* device_assignments;
	int* device_k;
	int* device_countPixels;

	float* pixelsInPtr = new float[pixels.size() * 3];

	for (int i = 0; i < pixels.size(); i++) {
		for (int j = 0; j < 3; j++) {
			pixelsInPtr[3 * i] = pixels[i].r;
			pixelsInPtr[3 * i + 1] = pixels[i].g;
			pixelsInPtr[3 * i + 2] = pixels[i].b;
		}
	}

	float* centroidsPtr = new float[k * 3];
	for (int i = 0; i < k; i++) {
		auto rndPixel = pixels[rand() % pixels.size()];
		centroidsPtr[3 * i] = rndPixel.r;
		centroidsPtr[3 * i + 1] = rndPixel.g;
		centroidsPtr[3 * i + 2] = rndPixel.b;
	}

	int countPixels = pixels.size();

	hipMalloc((void**)&device_data, sizeof(float) * pixels.size() * 3);
	hipMalloc((void**)&device_centroids, sizeof(float) * k * 3);
	hipMalloc((void**)&device_assignments, sizeof(int) * pixels.size());
	hipMalloc((void**)&device_k, sizeof(int));
	hipMalloc((void**)&device_countPixels, sizeof(int));

	hipMemcpy(device_data, pixelsInPtr, sizeof(float) * pixels.size() * 3, hipMemcpyHostToDevice);
	hipMemcpy(device_centroids, centroidsPtr, sizeof(float) * k * 3, hipMemcpyHostToDevice);
	hipMemcpy(device_assignments, centroidsPtr, sizeof(float) * k * 3, hipMemcpyHostToDevice);
	hipMemcpy(device_k, &k, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_countPixels, &countPixels, sizeof(int), hipMemcpyHostToDevice);


	dim3 block_size = 256;
	dim3 grid_size = (pixels.size() + 256 - 1) / 256;


	kmeans_kernel << <grid_size, block_size >> > (device_data, device_centroids, device_assignments, device_k, device_countPixels);
	
	int* assignmentsPtr = new int[pixels.size()];
	hipMemcpy(assignmentsPtr, device_assignments, sizeof(int) * pixels.size(), hipMemcpyDeviceToHost);

	std::vector<Pixel> sums(k);
	std::vector<int> counts(k, 0);

	for (int i = 0; i < pixels.size(); i++) {
		auto a = assignmentsPtr[i];
		sums[a].r += pixels[i].r;
		sums[a].g += pixels[i].g;
		sums[a].b += pixels[i].b;
		counts[a]++;
	}

	std::vector<Pixel> centroids(k);
	for (int i = 0; i < k; i++) {
		if (counts[i] == 0) counts[i]++;
		centroids[i].r = sums[i].r / counts[i];
		centroids[i].g = sums[i].g / counts[i];
		centroids[i].b = sums[i].b / counts[i];
	}

	ClusteredImage image;

	auto vectorAssignments = std::vector<int>(pixels.size());

	for (size_t i = 0; i < pixels.size(); i++)
	{
		vectorAssignments[i] = assignmentsPtr[i];
	}

	image.assignments = vectorAssignments;
	image.centroids = centroids;

	hipFree(device_data);
	hipFree(device_assignments);
	hipFree(device_assignments);
	hipFree(device_countPixels);
	hipFree(device_k);

	return image;
}


__global__ void kernel_hc(bool* hc)
{
	*hc = true;
}


bool healthCheck()
{
	try
	{
		bool* hc = new bool;
		*hc = false;

		bool* device_hc;

		hipMalloc((void**)&device_hc, sizeof(bool));
		hipMemcpy(device_hc, &hc, sizeof(bool), hipMemcpyHostToDevice);


		kernel_hc << <1, 1 >> > (device_hc);
		hipMemcpy(hc, device_hc, sizeof(bool), hipMemcpyDeviceToHost);
		hipFree(device_hc);
		return *hc;
	}
	catch (const std::exception&)
	{
		return false;
	}
	

	
}


