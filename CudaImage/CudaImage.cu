#include "hip/hip_runtime.h"
#include "CudaImage.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../MagaFirstTermCourseProject.FormsApp/ClusteredImage.h"
#include <iostream>

using namespace CommonTypes;

/*
* DOCS
* d_labels == assignments
*/


__global__ void kmeans_kernel(float* d_data, float* d_centroids, int* d_labels, int k, int countPixels) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < countPixels) {
		float min_dist = sqrt((d_data[idx * 3] - d_centroids[0]) * (d_data[idx * 3] - d_centroids[0])
							   + (d_data[idx * 3 + 1] - d_centroids[1]) * (d_data[idx * 3 + 1] - d_centroids[1])
								   + (d_data[idx * 3 + 2] - d_centroids[2]) * (d_data[idx * 3 + 2] - d_centroids[2]));
		int label = 0;

		for (int i = 0; i < k; i++) {
			float dist = 0.0f;

			dist = sqrt((d_data[idx * 3] - d_centroids[k * 3]) * (d_data[idx * 3] - d_centroids[k * 3])
				+ (d_data[idx * 3 + 1] - d_centroids[k * 3 + 1]) * (d_data[idx * 3 + 1] - d_centroids[k * 3 + 1])
				+ (d_data[idx * 3 + 2] - d_centroids[k * 3 + 2]) * (d_data[idx * 3 + 2] - d_centroids[k * 3 + 2]));

			if (dist < min_dist) {
				min_dist = dist;
				label = i;
			}
		}

		d_labels[idx] = label;
	}
}



ClusteredImage calc(std::vector<Pixel>& pixels, int k)
{
	float* device_data;
	float* device_centroids;
	int* device_assignments;

	float* pixelsInPtr = new float[pixels.size() * 3];

	for (int i = 0; i < pixels.size(); i++) {
		for (int j = 0; j < 3; j++) {
			pixelsInPtr[3 * i] = pixels[i].r;
			pixelsInPtr[3 * i + 1] = pixels[i].g;
			pixelsInPtr[3 * i + 2] = pixels[i].b;
		}
	}

	float* centroidsPtr = new float[k * 3];
	for (int i = 0; i < k; i++) {
		auto rndPixel = pixels[rand() % pixels.size()];
		centroidsPtr[3 * i] = rndPixel.r;
		centroidsPtr[3 * i + 1] = rndPixel.g;
		centroidsPtr[3 * i + 2] = rndPixel.b;
	}

	hipMalloc((void**)&device_data, sizeof(float) * pixels.size() * 3);
	hipMalloc((void**)&device_centroids, sizeof(float) * k * 3);
	hipMalloc((void**)&device_assignments, sizeof(int) * pixels.size());

	hipMemcpy(device_data, pixelsInPtr, sizeof(float) * pixels.size() * 3, hipMemcpyHostToDevice);
	hipMemcpy(device_centroids, centroidsPtr, sizeof(float) * k * 3, hipMemcpyHostToDevice);
	hipMemcpy(device_assignments, centroidsPtr, sizeof(float) * k * 3, hipMemcpyHostToDevice);


	dim3 block_size = 256;
	dim3 grid_size = (pixels.size() + 256 - 1) / 256;

	kmeans_kernel <<<grid_size, block_size >>> (device_data, device_centroids, device_assignments, k, pixels.size());

	int* assignmentsPtr = new int[pixels.size()];
    hipMemcpy(assignmentsPtr, device_assignments, sizeof(int) * pixels.size(), hipMemcpyDeviceToHost);

	


	std::vector<Pixel> sums(k);
	std::vector<int> counts(k, 0);

	for (int i = 0; i < pixels.size(); i++) {
		auto a = assignmentsPtr[i];
		sums[a].r += pixels[i].r;
		sums[a].g += pixels[i].g;
		sums[a].b += pixels[i].b;
		counts[a]++;
	}

	std::vector<Pixel> centroids(k);
	for (int i = 0; i < k; i++) {
		if (counts[i] == 0) counts[i]++;
		centroids[i].r = sums[i].r / counts[i];
		centroids[i].g = sums[i].g / counts[i];
		centroids[i].b = sums[i].b / counts[i];
	}

	ClusteredImage image;

	//image.assignments = assignments;
	image.centroids = centroids;

	hipFree(device_data);
	hipFree(device_assignments);
	hipFree(device_assignments);

	return image;
}