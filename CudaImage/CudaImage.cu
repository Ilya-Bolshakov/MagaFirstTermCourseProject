#include "hip/hip_runtime.h"
#include "CudaImage.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../MagaFirstTermCourseProject.FormsApp/ClusteredImage.h"

using namespace CommonTypes;




__global__ void kmeans_kernel(float* d_data, float* d_centroids, int* d_labels, int k, int num_pixels, int num_channels) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < num_pixels) {
		float min_dist = INFINITY;
		int label = 0;

		for (int i = 0; i < k; i++) {
			float dist = 0.0f;
			/*for (int j = 0; j < num_channels; j++) {
				float diff = d_data[idx * num_channels + j] - d_centroids[i * num_channels + j];
				dist += diff * diff;
			}*/

			double dist = distance(pixels[i], centroids[j]);

			if (dist < min_dist) {
				min_dist = dist;
				label = i;
			}
		}

		d_labels[idx] = label;
	}
}


ClusteredImage calc(std::vector<Pixel>& pixels, int k)
{
	std::vector<Pixel> centroids(k);
	for (int i = 0; i < k; i++) {
		centroids[i] = pixels[rand() % pixels.size()];
	}


	ClusteredImage c;

	Pixel p;

	p.b = 5;
	p.r = 10;
	p.g = 50;

	c.centroids.push_back(p);

	//unsigned* dev_srcImage;
	//size_t size = sizeof(unsigned) * width * height;
	//if (hipMalloc((void**)&dev_srcImage, size) !=
	//	hipError_t::hipSuccess) return c;
	//if (hipMemcpy(dev_srcImage, srcImage, size,
	//	hipMemcpyHostToDevice) != hipError_t::hipSuccess)
	//	return c;
	//// ������������ ���������� ����� �� ���� �����
	//	dim3 threads(128, 128);
	//dim3 blocks((width + threads.x - 1) / threads.x,
	//	(height + threads.y - 1) / threads.y);
	//kernel << <threads, blocks >> > (dev_srcImage, width,
	//	height);
	//hipError_t error = hipMemcpy(srcImage, dev_srcImage,
	//	size,
	//	hipMemcpyDeviceToHost);
	//if (error != hipError_t::hipSuccess) return c;
	//hipFree(dev_srcImage);




	return c;
}