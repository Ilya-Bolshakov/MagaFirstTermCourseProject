#include "hip/hip_runtime.h"
#include "CudaImage.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../MagaFirstTermCourseProject.FormsApp/ClusteredImage.h"
#include <thrust/device_vector.h>

using namespace CommonTypes;

/*
* DOCS
* d_labels == assignments
*/


__global__ void kmeans_kernel(std::vector<Pixel> centroids, int* assignments, int k, std::vector<Pixel> pixels) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < pixels.size()) {
		float minDist = INFINITY;
		int label = 0;

		for (int i = 0; i < k; i++) {
			float dist = 0.0f;
			
			dist = sqrt(pow(pixels[idx].r - centroids[i].r, 2) + pow(pixels[idx].g - centroids[i].g, 2) + pow(pixels[idx].b - centroids[i].b, 2));

			if (dist < minDist) {
				minDist = dist;
				label = i;
			}
		}

		assignments[idx] = label;
	}
}




ClusteredImage calc(std::vector<Pixel>& pixels, int k)
{
	std::vector<Pixel> centroids(k);
	thrust::device_vector<int> assignments;

	for (int i = 0; i < k; i++) {
		centroids[i] = pixels[rand() % pixels.size()];
	}

	//hipMalloc((void**)&d_pixels, sizeof(Pixel) * pixels.size());
	//hipMalloc((void**)&centroids, sizeof(Pixel) * centroids.size());
	hipMalloc((void**)&assignments, sizeof(int) * pixels.size());

	thrust::device_vector<Pixel> d_pixels(pixels);
	thrust::device_vector<Pixel> d_centroids(centroids);

	int block_size = 256;
	int grid_size = (pixels.size() + block_size - 1) / block_size;

	kmeans_kernel <<<grid_size, block_size >>> (d_pixels, d_centroids, assignments, k);


	std::vector<Pixel> sums(k);
	std::vector<int> counts(k, 0);
	for (int i = 0; i < pixels.size(); i++) {
		sums[assignments[i]].r += pixels[i].r;
		sums[assignments[i]].g += pixels[i].g;
		sums[assignments[i]].b += pixels[i].b;
		counts[assignments[i]]++;
	}
	for (int i = 0; i < k; i++) {
		if (counts[i] == 0) counts[i]++;
		centroids[i].r = sums[i].r / counts[i];
		centroids[i].g = sums[i].g / counts[i];
		centroids[i].b = sums[i].b / counts[i];
	}

	ClusteredImage image;

	std::vector<int> a;

	for (size_t i = 0; i < assignments.size(); i++)
	{
		a.push_back(assignments[i]);
	}

	image.assignments = a;
	image.centroids = centroids;

	return image;
}